#include "hip/hip_runtime.h"
// **************************************************************************
//                                   eam.cu
//                             -------------------
//                   Trung Dac Nguyen, W. Michael Brown (ORNL)
//
//  Device code for acceleration of the eam pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                : 
//    email                : brownw@ornl.gov nguyentd@ornl.gov
// ***************************************************************************/

#ifdef NV_KERNEL
#include "lal_aux_fun1.h"
texture<float4> pos_tex;
texture<float> fp_tex;

texture<float4> rhor_sp1_tex;
texture<float4> rhor_sp2_tex;
texture<float4> frho_sp1_tex;
texture<float4> frho_sp2_tex;
texture<float4> z2r_sp1_tex;
texture<float4> z2r_sp2_tex;

#ifdef _DOUBLE_DOUBLE
ucl_inline double4 fetch_rhor_sp1(const int& i, const double4 *rhor_spline1) { 
  return rhor_spline1[i]; 
}
ucl_inline double4 fetch_rhor_sp2(const int& i, const double4 *rhor_spline2) { 
  return rhor_spline2[i]; 
}
ucl_inline double4 fetch_frho_sp1(const int& i, const double4 *frho_spline1) { 
  return frho_spline1[i]; 
}
ucl_inline double4 fetch_frho_sp2(const int& i, const double4 *frho_spline2) { 
  return frho_spline2[i]; 
}
ucl_inline double4 fetch_z2r_sp1(const int& i, const double4 *z2r_spline1) { 
  return z2r_spline1[i]; 
}
ucl_inline double4 fetch_z2r_sp2(const int& i, const double4 *z2r_spline2) { 
  return z2r_spline2[i]; 
}
#endif

#ifndef _DOUBLE_DOUBLE
ucl_inline float4 fetch_pos(const int& i, const float4 *pos)
  { return tex1Dfetch(pos_tex, i); }
ucl_inline float fetch_q(const int& i, const float *fp) 
  { return tex1Dfetch(fp_tex, i); }

ucl_inline float4 fetch_rhor_sp1(const int& i, const float4 *rhor_spline1) 
  { return tex1Dfetch(rhor_sp1_tex, i); }
ucl_inline float4 fetch_rhor_sp2(const int& i, const float4 *rhor_spline2) 
  { return tex1Dfetch(rhor_sp2_tex, i); }
ucl_inline float4 fetch_frho_sp1(const int& i, const float4 *frho_spline1) 
  { return tex1Dfetch(frho_sp1_tex, i); }
ucl_inline float4 fetch_frho_sp2(const int& i, const float4 *frho_spline2) 
  { return tex1Dfetch(frho_sp2_tex, i); }
ucl_inline float4 fetch_z2r_sp1(const int& i, const float4 *z2r_spline1) 
  { return tex1Dfetch(z2r_sp1_tex, i); }
ucl_inline float4 fetch_z2r_sp2(const int& i, const float4 *z2r_spline2) 
  { return tex1Dfetch(z2r_sp2_tex, i); }
#endif

#else // OPENCL

#define fetch_q(i,y) fp_[i]
#define fetch_rhor_sp1(i,y) rhor_spline1[i]
#define fetch_rhor_sp2(i,y) rhor_spline2[i]
#define fetch_frho_sp1(i,y) frho_spline1[i]
#define fetch_frho_sp2(i,y) frho_spline2[i]
#define fetch_z2r_sp1(i,y) z2r_spline1[i] 
#define fetch_z2r_sp2(i,y) z2r_spline2[i]

#endif

#define MIN(A,B) ((A) < (B) ? (A) : (B))
#define MAX(A,B) ((A) > (B) ? (A) : (B))

#define store_energy_fp(rho,energy,ii,inum,tid,t_per_atom,offset,           \
                        eflag,vflag,engv,rdrho,nrho,i)                      \
  if (t_per_atom>1) {                                                       \
    __local acctyp red_acc[BLOCK_PAIR];                                     \
    red_acc[tid]=rho;                                                       \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      if (offset < s)                                                       \
         red_acc[tid] += red_acc[tid+s];                                    \
      }                                                                     \
      rho=red_acc[tid];                                                     \
  }                                                                         \
  if (offset==0) {                                                          \
    numtyp p = rho*rdrho + (numtyp)1.0;                                     \
    int m=p;                                                                \
    m = MAX(1,MIN(m,nrho-1));                                               \
    p -= m;                                                                 \
    p = MIN(p,(numtyp)1.0);                                                 \
    int index = type2frho[itype]*(nrho+1)+m;                                \
    numtyp4 coeff = fetch_frho_sp1(index, frho_spline1);                    \
    numtyp fp = (coeff.x*p + coeff.y)*p + coeff.z;                          \
    fp_[i]=fp;                                                              \
    if (eflag>0) {                                                          \
      coeff = fetch_frho_sp2(index, frho_spline2);                          \
      energy = ((coeff.x*p + coeff.y)*p + coeff.z)*p + coeff.w;             \
      engv[ii]=(acctyp)2.0*energy;                                          \
    }                                                                       \
  }

#define store_answers_eam(f, energy, virial, ii, inum, tid, t_per_atom,     \
                      offset, elag, vflag, ans, engv)                       \
  if (t_per_atom>1) {                                                       \
    __local acctyp red_acc[6][BLOCK_PAIR];                                  \
    red_acc[0][tid]=f.x;                                                    \
    red_acc[1][tid]=f.y;                                                    \
    red_acc[2][tid]=f.z;                                                    \
    red_acc[3][tid]=energy;                                                 \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      if (offset < s) {                                                     \
        for (int r=0; r<4; r++)                                             \
          red_acc[r][tid] += red_acc[r][tid+s];                             \
      }                                                                     \
    }                                                                       \
    f.x=red_acc[0][tid];                                                    \
    f.y=red_acc[1][tid];                                                    \
    f.z=red_acc[2][tid];                                                    \
    energy=red_acc[3][tid];                                                 \
    if (vflag>0) {                                                          \
      for (int r=0; r<6; r++)                                               \
        red_acc[r][tid]=virial[r];                                          \
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                       \
        if (offset < s) {                                                   \
          for (int r=0; r<6; r++)                                           \
            red_acc[r][tid] += red_acc[r][tid+s];                           \
        }                                                                   \
      }                                                                     \
      for (int r=0; r<6; r++)                                               \
        virial[r]=red_acc[r][tid];                                          \
    }                                                                       \
  }                                                                         \
  if (offset==0) {                                                          \
    if (eflag>0) {                                                          \
      engv[ii]+=energy;                                                     \
      engv+=inum;                                                           \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (int i=0; i<6; i++) {                                             \
        engv[ii]=virial[i];                                                 \
        engv+=inum;                                                         \
      }                                                                     \
    }                                                                       \
    ans[ii]=f;                                                              \
  }

__kernel void kernel_energy(__global numtyp4 *x_, __global int2 *type2rhor_z2r,
                            __global int *type2frho, 
                            __global numtyp4 *rhor_spline2, 
                            __global numtyp4 *frho_spline1,
                            __global numtyp4 *frho_spline2,
                            __global int *dev_nbor, __global int *dev_packed,
                            __global numtyp *fp_, __global acctyp *engv, 
                            const int eflag, const int inum, 
                            const int nbor_pitch, const int ntypes, 
                            const numtyp cutforcesq, const numtyp rdr, 
                            const numtyp rdrho, const int nrho, const int nr,
                            const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);
  
  acctyp rho = (acctyp)0;
  acctyp energy = (acctyp)0;
   
  if (ii<inum) {
    __global int *nbor, *list_end;
    int i, numj, n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,list_end,nbor);
  
    numtyp4 ix=fetch_pos(i,x_); //x_[i];
    int itype=ix.w;
    
    for ( ; nbor<list_end; nbor+=n_stride) {
      int j=*nbor;
      j &= NEIGHMASK;

      numtyp4 jx=fetch_pos(j,x_); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;
      
      if (rsq<cutforcesq) {
        numtyp p = ucl_sqrt(rsq)*rdr + (numtyp)1.0;
        int m=p;
        m = MIN(m,nr-1);
        p -= m;
        p = MIN(p,(numtyp)1.0);
        
        int mtype = jtype*ntypes+itype;
        int index = type2rhor_z2r[mtype].x*(nr+1)+m;
        numtyp4 coeff = fetch_rhor_sp2(index, rhor_spline2);
        rho += ((coeff.x*p + coeff.y)*p + coeff.z)*p + coeff.w;
      }
    } // for nbor
    
    store_energy_fp(rho,energy,ii,inum,tid,t_per_atom,offset,
        eflag,vflag,engv,rdrho,nrho,i);
  } // if ii
}

__kernel void kernel_energy_fast(__global numtyp4 *x_, 
                                 __global int2 *type2rhor_z2r_in,
                                 __global int *type2frho_in, 
                                 __global numtyp4 *rhor_spline2, 
                                 __global numtyp4 *frho_spline1,
                                 __global numtyp4 *frho_spline2,
                                 __global int *dev_nbor, 
                                 __global int *dev_packed, __global numtyp *fp_, 
                                 __global acctyp *engv, const int eflag, 
                                 const int inum, const int nbor_pitch,
                                 const int ntypes, const numtyp cutforcesq, 
                                 const numtyp rdr, const numtyp rdrho,
                                 const int nrho, const int nr, 
                                 const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);
  
  __local int2 type2rhor_z2r[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local int type2frho[MAX_SHARED_TYPES];

  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    type2rhor_z2r[tid]=type2rhor_z2r_in[tid];
  }
  
  if (tid<MAX_SHARED_TYPES) {
    type2frho[tid]=type2frho_in[tid];
  }

  acctyp rho = (acctyp)0;
  acctyp energy = (acctyp)0;
  
  __syncthreads(); 

  if (ii<inum) {
    __global int *nbor, *list_end;
    int i, numj, n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,list_end,nbor);
  
    numtyp4 ix=fetch_pos(i,x_); //x_[i];
    int itype=ix.w;
    
    for ( ; nbor<list_end; nbor+=n_stride) {
      int j=*nbor;
      j &= NEIGHMASK;

      numtyp4 jx=fetch_pos(j,x_); //x_[j];

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;
      
      if (rsq<cutforcesq) {
        numtyp p = ucl_sqrt(rsq)*rdr + (numtyp)1.0;
        int m=p;
        m = MIN(m,nr-1);
        p -= m;
        p = MIN(p,(numtyp)1.0);
        
        int jtype=fast_mul((int)MAX_SHARED_TYPES,jx.w);
        int mtype = jtype+itype;
        int index = type2rhor_z2r[mtype].x*(nr+1)+m;
        numtyp4 coeff = fetch_rhor_sp2(index, rhor_spline2);
        rho += ((coeff.x*p + coeff.y)*p + coeff.z)*p + coeff.w;
      }
    } // for nbor
    
    store_energy_fp(rho,energy,ii,inum,tid,t_per_atom,offset,
                    eflag,vflag,engv,rdrho,nrho,i);
  } // if ii
}

__kernel void kernel_pair(__global numtyp4 *x_, __global numtyp *fp_,
                          __global int2 *type2rhor_z2r,
                          __global numtyp4 *rhor_spline1, 
                          __global numtyp4 *z2r_spline1,
                          __global numtyp4 *z2r_spline2,
                          __global int *dev_nbor, __global int *dev_packed, 
                          __global acctyp4 *ans, __global acctyp *engv, 
                          const int eflag, const int vflag, 
                          const int inum, const int nbor_pitch,
                          const int ntypes, const numtyp cutforcesq, 
                          const numtyp rdr, const int nr,
                          const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0;
  f.y=(acctyp)0;
  f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;
  
  if (ii<inum) {
    __global int *nbor, *list_end;
    int i, numj, n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,list_end,nbor);
  
    numtyp4 ix=fetch_pos(i,x_); //x_[i];
    numtyp ifp=fetch_q(i,fp_);  //fp_[i];
    int itype=ix.w;

    for ( ; nbor<list_end; nbor+=n_stride) {
      int j=*nbor;
      j &= NEIGHMASK;

      numtyp4 jx=fetch_pos(j,x_); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;
      
      if (rsq<cutforcesq) {
        numtyp r = ucl_sqrt(rsq);
        numtyp p = r*rdr + (numtyp)1.0;
        int m=p;
        m = MIN(m,nr-1);
        p -= m;
        p = MIN(p,(numtyp)1.0);
        
        int mtype,index;
        numtyp4 coeff;

        mtype = itype*ntypes+jtype;
        index = type2rhor_z2r[mtype].x*(nr+1)+m;
        coeff = fetch_rhor_sp1(index, rhor_spline1); 
        numtyp rhoip = (coeff.x*p + coeff.y)*p + coeff.z;

        mtype = jtype*ntypes+itype;
        index = type2rhor_z2r[mtype].x*(nr+1)+m;
        coeff = fetch_rhor_sp1(index, rhor_spline1); 
        numtyp rhojp = (coeff.x*p + coeff.y)*p + coeff.z;
              
        mtype = itype*ntypes+jtype;
        index = type2rhor_z2r[mtype].y*(nr+1)+m;
        coeff = fetch_z2r_sp1(index, z2r_spline1);
        numtyp z2p = (coeff.x*p + coeff.y)*p + coeff.z;
        coeff = fetch_z2r_sp2(index, z2r_spline2);
        numtyp z2 = ((coeff.x*p + coeff.y)*p + coeff.z)*p + coeff.w;
        
        numtyp recip = ucl_recip(r);
        numtyp phi = z2*recip;
        numtyp phip = z2p*recip - phi*recip;
        numtyp psip = ifp*rhojp + fetch_q(j,fp_)*rhoip + phip; 
        numtyp force = -psip*recip;
        
        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          energy += phi;
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }
    } // for nbor
    store_answers_eam(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii

}

__kernel void kernel_pair_fast(__global numtyp4 *x_, __global numtyp *fp_,
                          __global int2 *type2rhor_z2r_in,
                          __global numtyp4 *rhor_spline1, 
                          __global numtyp4 *z2r_spline1,
                          __global numtyp4 *z2r_spline2,
                          __global int *dev_nbor, __global int *dev_packed, 
                          __global acctyp4 *ans, __global acctyp *engv, 
                          const int eflag, const int vflag, const int inum, 
                          const int nbor_pitch,
                          const numtyp cutforcesq, 
                          const numtyp rdr, const int nr,
                          const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);
  
  __local int2 type2rhor_z2r[MAX_SHARED_TYPES*MAX_SHARED_TYPES];

  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    type2rhor_z2r[tid]=type2rhor_z2r_in[tid];
  }

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    __global int *nbor, *list_end;
    int i, numj, n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,list_end,nbor);

    numtyp4 ix=fetch_pos(i,x_); //x_[i];
    numtyp ifp=fetch_q(i,fp_); //fp_[i];
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    for ( ; nbor<list_end; nbor+=n_stride) {
      int j=*nbor;
      j &= NEIGHMASK;

      numtyp4 jx=fetch_pos(j,x_); //x_[j];
      int jw=jx.w;
      int jtype=fast_mul((int)MAX_SHARED_TYPES,jw);
      
      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;
        
      if (rsq<cutforcesq) {
        numtyp r = ucl_sqrt(rsq);
        numtyp p = r*rdr + (numtyp)1.0;
        int m=p;
        m = MIN(m,nr-1);
        p -= m;
        p = MIN(p,(numtyp)1.0);
        
        numtyp4 coeff;
        int mtype,index;
        
        mtype = itype+jw;
        index = type2rhor_z2r[mtype].x*(nr+1)+m;
        coeff = fetch_rhor_sp1(index, rhor_spline1); 
        numtyp rhoip = (coeff.x*p + coeff.y)*p + coeff.z;
        
        mtype = jtype+iw;
        index = type2rhor_z2r[mtype].x*(nr+1)+m;
        coeff = fetch_rhor_sp1(index, rhor_spline1); 
        numtyp rhojp = (coeff.x*p + coeff.y)*p + coeff.z;
        
        mtype = itype+jw;
        index = type2rhor_z2r[mtype].y*(nr+1)+m;
        coeff = fetch_z2r_sp1(index, z2r_spline1);
        numtyp z2p = (coeff.x*p + coeff.y)*p + coeff.z;
        coeff = fetch_z2r_sp2(index, z2r_spline2);
        numtyp z2 = ((coeff.x*p + coeff.y)*p + coeff.z)*p + coeff.w;
      
        numtyp recip = ucl_recip(r);
        numtyp phi = z2*recip;
        numtyp phip = z2p*recip - phi*recip;
        numtyp psip = ifp*rhojp + fetch_q(j,fp_)*rhoip + phip;
        numtyp force = -psip*recip;
        
        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          energy += phi;
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }
    } // for nbor
    store_answers_eam(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii
}

